#include "hip/hip_runtime.h"
#include "material.cuh"

__device__ Material::Material()
{
}

__device__ Material::~Material()
{
}

__device__ bool Material::scatter(const Ray& rayIn, const HitRecord& rec, vec3& attenuation, Ray& scattered) const
{
    attenuation = vec3(1.0, 1.0, 1.0);
    scattered = Ray(vec3(0.5,0.1,0.6), vec3(0.1,0.2,0.5));
    return true;
}

__device__ Lambertian::Lambertian(const vec3& albedo)
    : m_albedo(albedo)
{
}

__device__ bool Lambertian::scatter(const Ray& rayIn, const HitRecord& rec, vec3& attenuation, Ray& scattered) const
{
    vec3 scatterDir = rec.normal + randomNormalizeVec3();
    if (nearZero(scatterDir))
        scatterDir = rec.normal;
    scatterDir = unit_vector(scatterDir);

    scattered = Ray(rec.pos, scatterDir);
    attenuation = m_albedo;

    return true;
}

__device__ Metal::Metal(const vec3& albedo, double fuzz)
    : m_albedo(albedo),
      m_fuzz(fuzz)
{
}

__device__ bool Metal::scatter(const Ray& rayIn, const HitRecord& rec, vec3& attenuation, Ray& scattered) const
{
    vec3 reflected = reflect(rayIn.getDirection(), rec.normal) + m_fuzz * randomNormalizeVec3();
    reflected = unit_vector(reflected);

    scattered = Ray(rec.pos, reflected);
    attenuation = m_albedo;

    return true;
}

__device__ Dielectric::Dielectric(double ratio)
    : m_ratio(ratio)
{
}

__device__ bool Dielectric::scatter(const Ray& rayIn, const HitRecord& rec, vec3& attenuation, Ray& scattered) const
{
    attenuation = vec3(1.0, 1.0, 1.0);
    double ri = rec.frontFace ? (1.0 / m_ratio) : m_ratio;

    vec3 unit_direction = unit_vector(rayIn.getDirection());
    double cos_theta = fmin(dot(-unit_direction, rec.normal), 1.0);
    double sin_theta = sqrt(1.0 - cos_theta * cos_theta);

    bool cannot_refract = ri * sin_theta > 1.0;
    vec3 direction;

    if (cannot_refract || reflectance(cos_theta, ri) > randomDouble())
        direction = reflect(unit_direction, rec.normal);
    else
        direction = refract(unit_direction, rec.normal, ri);

    scattered = Ray(rec.pos, direction);
    return true;
}

__device__ double Dielectric::reflectance(double cosine, double refraction_index)
{
    auto r0 = (1 - refraction_index) / (1 + refraction_index);
    r0 = r0 * r0;
    return r0 + (1 - r0) * std::pow((1 - cosine), 5);
}
