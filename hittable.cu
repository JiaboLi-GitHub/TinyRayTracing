#include "hip/hip_runtime.h"
#include "hittable.cuh"

__device__ Hittable::Hittable()
{
}

__device__ Hittable::~Hittable()
{
}

__device__ HittableList::HittableList(Hittable** hittables, size_t size)
    : m_hittables(hittables), m_size(size)
{
}

__device__ bool HittableList::hit(const Ray& ray, Interval rayT, HitRecord& rec) const
{
    HitRecord tmpRec;
    bool hitAnything = false;
    double closestSoFar = rayT.getMax();

    for (int i = 0; i < m_size; ++i)
    {
        Hittable* hittable = m_hittables[i];
        if (hittable->hit(ray, Interval(rayT.getMin(), closestSoFar), tmpRec))
        {
            hitAnything = true;
            closestSoFar = tmpRec.t;
            rec = tmpRec;
        }
    }

    return hitAnything;
}

__device__ Sphere::Sphere(const vec3& center, double r, Material* material)
    : m_center(center), m_r(r), m_material(material)
{
}

__device__ Sphere::~Sphere()
{
}

__device__ bool Sphere::hit(const Ray& ray, Interval rayT, HitRecord& rec) const
{
    vec3 oc = m_center - ray.getOrigin();
    double a = dot(ray.getDirection(), ray.getDirection());
    double b = -2.0 * dot(ray.getDirection(), oc);
    double c = dot(oc, oc) - m_r * m_r;
    double discriminant = b * b - 4 * a * c;

    if (discriminant < 0)
    {
        return false;
    }

    // 计算两个 t 值
    double sqrtDiscriminant = sqrt(discriminant);
    double t1 = (-b - sqrtDiscriminant) / (2.0 * a);
    double t2 = (-b + sqrtDiscriminant) / (2.0 * a);

    // 找到第一个有效的交点
    if (t1 < rayT.getMin() || t1 > rayT.getMax())
    {
        t1 = t2; // 尝试第二个交点
        if (t1 < rayT.getMin() || t1 > rayT.getMax())
        {
            return false; // 两个交点都不在范围内
        }
    }

    rec.t = t1;
    rec.pos = ray.at(t1);

    //法线始终与光线方向相反
    vec3 normal = unit_vector(rec.pos - m_center);
    rec.frontFace = dot(ray.getDirection(), normal) < 0;
    if (rec.frontFace)
        rec.normal = normal;
    else
        rec.normal = -normal;
    rec.material = m_material;

    return true;
}
