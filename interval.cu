#include "hip/hip_runtime.h"
#include "interval.cuh"

__device__ Interval::Interval(double min, double max)
    : m_min(min), m_max(max)
{
}

__device__ Interval::~Interval()
{
}

__device__ double Interval::size() const
{
    return m_max - m_min;
}

__device__ bool Interval::contains(double x) const
{
    return m_min <= x && x <= m_max;
}

__device__ bool Interval::surrounds(double x) const
{
    return m_min < x && x < m_max;
}

__device__ double Interval::clamp(double x) const
{
    if (x < m_min) return m_min;
    if (x > m_max) return m_max;
    return x;
}

__device__ double Interval::getMin() const
{
    return m_min;
}

__device__ double Interval::getMax() const
{
    return m_max;
}
