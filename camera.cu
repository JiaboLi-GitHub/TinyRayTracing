#include "hip/hip_runtime.h"
#include "camera.cuh"
#include <hip/hip_runtime.h>
#include "material.cuh"

__device__ Camera::Camera(vec3* image)
    : m_image(image)
{
    init();
}

__device__ void Camera::render(Hittable* scene, int x, int y)
{
    vec3 color = vec3(0, 0, 0);
    for (int i = 0; i < m_samplesPerPixel; ++i)
    {
        Ray ray = getRay(x, y);
        color += rayColor(ray, scene);
    }
    color /= m_samplesPerPixel;

    m_image[x + y * ImageWidthD] = color;
    //m_image[x+y*ImageWidthD] = vec3(0.5, 0.8, 0.9);
    // printf("x = %d, y = %d:  %lf %lf %lf \n", x,y,color.r(),color.g(),color.b());
}

__device__ void Camera::init()
{
    m_cameraCenter = vec3(13, 2, 3);
    m_fov = 20;
    m_look = vec3(0, 0, 0);
    m_up = vec3(0, 1, 0);
    m_aspectRatio = AspectRatioD;
    m_imageWidth = ImageWidthD;
    m_pixelDeltaU = vec3(0, 0, 0);
    m_pixelDeltaV = vec3(0, 0, 0);
    m_pixel00Loc = vec3(0, 0, 0);
    m_samplesPerPixel = 10; // 每个像素的随机采样次数
    m_maxDepth = 5; // 最大递归深度
    m_defocusAngle = 0.6; // 光线通过屏幕的变化角度
    m_focusDist = 10.0; // 焦距

    m_imageHeight = int(ImageWidthD / AspectRatioD);
    double viewportHeight = 2.0 * m_focusDist * tan(toRadians(m_fov) / 2);
    double viewportWidth = viewportHeight * (double(m_imageWidth) / m_imageHeight);

    vec3 w, u, v;
    w = unit_vector(m_cameraCenter - m_look);
    u = unit_vector(cross(m_up, w));
    v = cross(w, u);

    vec3 viewPortU = viewportWidth * u;
    vec3 viewPortV = viewportHeight * v;

    m_pixelDeltaU = viewPortU / (m_imageWidth * 1.0);
    m_pixelDeltaV = viewPortV / (m_imageHeight * 1.0);

    auto pixel_upper_left = m_cameraCenter - m_focusDist * w - viewPortU / 2.0 - viewPortV / 2.0;
    m_pixel00Loc = pixel_upper_left + 0.5 * (m_pixelDeltaU + m_pixelDeltaV);

    auto defocusRadius = m_focusDist * tan(toRadians(m_defocusAngle / 2));
    m_defocusDiskU = u * defocusRadius;
    m_defocusDiskV = v * defocusRadius;
}

__device__ vec3 Camera::rayColor(const Ray& ray, const Hittable* scene)
{
    Ray currentRay = ray;
    vec3 attenuation(1.0, 1.0, 1.0); // 初始衰减
    vec3 finalColor(0.0, 0.0, 0.0); // 最终颜色
    int depth = 0;

    while (depth < m_maxDepth)
    {
        HitRecord rec;
        if (scene->hit(currentRay, Interval(0.001, InfinityD), rec))
        {
            Ray scattered;
            vec3 newAttenuation;

            // 处理散射
            if (rec.material->scatter(currentRay, rec, newAttenuation, scattered))
            {
                // 更新衰减和光线
                attenuation *= newAttenuation;
                currentRay = scattered;
                depth++;
            }
            else
            {
                // 如果没有散射，结束循环
                break;
            }
        }
        else
        {
            // 计算背景颜色
            double x = 0.5f * (currentRay.getDirection().y() + 1.0);
            vec3 backColor = (1.0 - x) * vec3(1.0, 1.0, 1.0) + x * vec3(0.5f, 0.7f, 1.0);
            finalColor += backColor * attenuation; // 加权背景颜色
            break; // 结束循环
        }
    }

    return finalColor;
}

__device__ Ray Camera::getRay(int i, int j) const
{
    vec3 offset = sampleSquare();

    vec3 pixelCenter = m_pixel00Loc + (i + offset.x()) * m_pixelDeltaU + (j + offset.y()) * m_pixelDeltaV;

    vec3 rayOrigin = (m_defocusAngle <= 0) ? m_cameraCenter : defocusDiskSample();
    vec3 rayDirection = pixelCenter - rayOrigin;

    rayDirection = unit_vector(rayDirection);

    return Ray(rayOrigin, rayDirection);
}

__device__ vec3 Camera::sampleSquare() const
{
    return vec3(randomDouble() - 0.5, randomDouble() - 0.5, 0);
}

__device__ vec3 Camera::defocusDiskSample() const
{
    vec3 vec3 = randomDisk();
    return m_cameraCenter + (vec3.x() * m_defocusDiskU) + (vec3.y() * m_defocusDiskV);
}
