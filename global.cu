#include "global.cuh"

double AspectRatio = 16.0 / 9.0;
int ImageWidth = 800;
int ImageHeight = ImageWidth / AspectRatio;
int ImageSize = ImageWidth * ImageHeight;
const dim3 BlockSize = dim3(16, 16);
const dim3 GridSize = dim3(ImageWidth / BlockSize.x + 1, ImageHeight / BlockSize.y + 1);

__device__ double AspectRatioD;
__device__ int ImageWidthD;
__device__ int ImageHeightD;
__device__ int ImageSizeD;
__device__ hiprandState* RandStateD = nullptr;
