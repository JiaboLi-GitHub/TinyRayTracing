#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <chrono>
#include "global.cuh"
#include "camera.cuh"
#include "hittable.cuh"
#include "material.cuh"

void initGlobal();

void flipImage(int width, int height, std::vector<unsigned char>& image_data);

__global__ void initRand();

__global__ void initCamera(Camera** camera, vec3* image);

__global__ void initScene(Hittable** scene);

__global__ void render(Camera** camera, Hittable** scene);

int main()
{
    // 获取开始时间
    auto start = std::chrono::high_resolution_clock::now();

    // 初始化全局数据
    initGlobal();

    // 初始化摄像机
    Camera** cameraD;
    vec3* imageD;

    hipMalloc((void**)&cameraD, sizeof(Camera*));
    hipMalloc((void**)&imageD, sizeof(vec3) * ImageSize);

    initCamera<<<1,1>>>(cameraD, imageD);
    hipDeviceSynchronize();

    // 初始化场景
    Hittable** sceneD;
    hipMalloc((void**)&sceneD, sizeof(Hittable*));
    initScene<<<1,1>>>(sceneD);

    // 渲染
    render<<<GridSize,BlockSize>>>(cameraD, sceneD);
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // 将图片从设备内存拷贝到主机内存
    std::vector<vec3> imageH(ImageSize);
    hipMemcpy(imageH.data(), imageD, ImageSize * sizeof(vec3), hipMemcpyDeviceToHost);

    // 保存图片
    std::vector<unsigned char> stbImage;
    stbImage.reserve(ImageWidth * ImageHeight * 3);
    for (const auto& v : imageH)
    {
        // std::cout << v << std::endl;
        stbImage.push_back(static_cast<unsigned char>(v.r() * 255));
        stbImage.push_back(static_cast<unsigned char>(v.g() * 255));
        stbImage.push_back(static_cast<unsigned char>(v.b() * 255));
    }
    flipImage(ImageWidth, ImageHeight, stbImage);
    if (stbi_write_png("output.png", ImageWidth, ImageHeight, 3, stbImage.data(), ImageWidth * 3))
    {
        std::cout << "图片保存成功" << std::endl;
    }
    else
    {
        std::cerr << "图片保存失败" << std::endl;
    }

    // 获取结束时间
    auto end = std::chrono::high_resolution_clock::now();

    // 计算持续时间
    std::chrono::duration<double, std::milli> duration = end - start;

    std::cout << "光线追踪耗时: " << duration.count() << " ms" << std::endl;

    return 0;
}

void initGlobal()
{
    hipMemcpyToSymbol(HIP_SYMBOL(AspectRatioD), &AspectRatio, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(ImageWidthD), &ImageWidth, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ImageHeightD), &ImageHeight, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ImageSizeD), &ImageSize, sizeof(int));

    hiprandState* randState;
    hipMalloc((void**)&randState, ImageSize * sizeof(hiprandState));
    hipMemcpyToSymbol(HIP_SYMBOL(RandStateD), &randState, sizeof(hiprandState*));

    initRand<<<GridSize,BlockSize>>>();
    hipDeviceSynchronize();
}

void flipImage(int width, int height, std::vector<unsigned char>& image_data)
{
    int bytes_per_pixel = 3;
    for (int y = 0; y < height / 2; ++y) {
        for (int x = 0; x < width; ++x) {
            int idx1 = (y * width + x) * bytes_per_pixel;
            int idx2 = ((height - 1 - y) * width + x) * bytes_per_pixel;

            // 交换行
            for (int b = 0; b < bytes_per_pixel; ++b) {
                std::swap(image_data[idx1 + b], image_data[idx2 + b]);
            }
        }
    }
}

__global__ void initRand()
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= ImageWidthD || y >= ImageHeightD)
        return;

    auto index = x + y * ImageWidthD;
    hiprand_init(1984 + index, 0, 0, &RandStateD[index]);
}

__global__ void initCamera(Camera** camera, vec3* image)
{
    *camera = new Camera(image);

    printf("%lf\n", AspectRatioD);
    printf("%d\n", ImageWidthD);
    printf("%d\n", ImageHeightD);
    printf("%d\n", ImageSizeD);
}

__global__ void initScene(Hittable** scene)
{
    Hittable** hittables = new Hittable*[1 + 22 * 22 + 3];
    int index = 0;

    auto ground_material = new Lambertian(vec3(0.5, 0.5, 0.5));
    hittables[index++] = new Sphere(vec3(0, -1000, 0), 1000, ground_material);

    for (int a = -11; a < 11; a++)
    {
        for (int b = -11; b < 11; b++)
        {
            auto choose_mat = randomDouble();
            vec3 center(a + 0.9 * randomDouble(), 0.2, b + 0.9 * randomDouble());

            if ((center - vec3(4, 0.2, 0)).length() > 0.9)
            {
                Material* sphere_material;

                if (choose_mat < 0.8)
                {
                    auto albedo = randomVec3(0.5, 1);
                    sphere_material = new Lambertian(albedo);
                    hittables[index++] = new Sphere(center, 0.2, sphere_material);
                }
                else if (choose_mat < 0.95)
                {
                    auto albedo = randomVec3(0.5, 1);
                    auto fuzz = randomDouble(0, 0.5);
                    sphere_material = new Metal(albedo, fuzz);
                    hittables[index++] = new Sphere(center, 0.2, sphere_material);
                }
                else
                {
                    sphere_material = new Dielectric(1.5);
                    hittables[index++] = new Sphere(center, 0.2, sphere_material);
                }
            }
        }
    }

    auto material1 = new Dielectric(1.5);
    hittables[index++] = new Sphere(vec3(0.0, 1.0, 0.0), 1.0, material1);
    auto material1_1 = new Dielectric(1.00 / 1.50);
    hittables[index++] = new Sphere(vec3(0.0, 1.0, 0.0), 0.8, material1_1);

    auto material2 = new Lambertian(vec3(0.4, 0.2, 0.1));
    hittables[index++] =  new Sphere(vec3(-4.0, 1.0, 0.0), 1.0, material2);

    auto material3 = new Metal(vec3(0.7, 0.6, 0.5), 0.0);
    hittables[index++] =  new Sphere(vec3(4.0, 1.0, 0.0), 1.0, material3);

    *scene = new HittableList(hittables, index);

    printf("initScene OK\n");
}

__global__ void render(Camera** camera, Hittable** scene)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= ImageWidthD || y >= ImageHeightD)
        return;

    (*camera)->render(*scene, x, y);
}

